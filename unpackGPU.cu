
#include <hip/hip_runtime.h>
#ifdef USE_GPU
#include "unpackGPU.cuh"
#include "cuda_rt_call.h"

#include <cuda_runtime.h>
#include <cub/util_debug.cuh>

#include <iostream>
#include <cassert>

constexpr auto kStripsPerChannel = SiStripConditionsBase::kStripsPerChannel;

__global__
static void unpackChannels(const ChannelLocsBase* chanlocs, const SiStripConditionsGPU* conditions,
                           uint8_t* alldata, detId_t* detId, stripId_t* stripId,
                           float* noise, float* gain, bool* bad)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;

  const auto chan = nthreads*bid + tid;
  if (chan < chanlocs->size()) {
    const auto fedid = chanlocs->fedID(chan);
    const auto fedch = chanlocs->fedCh(chan);
    const auto detid = conditions->detID(fedid, fedch);
    const auto ipoff = kStripsPerChannel*conditions->iPair(fedid, fedch);

    const auto data = chanlocs->input(chan);
    const auto len = chanlocs->length(chan);

    if (data != nullptr && len > 0) {
      auto aoff = chanlocs->offset(chan);
      auto choff = chanlocs->inoff(chan);
      const auto end = aoff + len;

      while (aoff < end) {
        stripId[aoff] = invStrip;
        detId[aoff] = detid;
        alldata[aoff] = data[(choff++)^7];
        auto stripIndex = alldata[aoff++] + ipoff;
 
        stripId[aoff] = invStrip;
        detId[aoff] = detid;
        alldata[aoff] = data[(choff++)^7];
        const auto groupLength = alldata[aoff++];

        for (auto i = 0; i < groupLength; ++i) {
          noise[aoff] = conditions->noise(fedid, fedch, stripIndex);
          gain[aoff]  = conditions->gain(fedid, fedch, stripIndex);
          bad[aoff]   = conditions->bad(fedid, fedch, stripIndex);
          detId[aoff] = detid;
          stripId[aoff] = stripIndex++;
          alldata[aoff++] = data[(choff++)^7];
        }
      }
    }
  }
}

void unpackChannelsGPU(const ChannelLocsGPU& chanlocs, const SiStripConditionsGPU* conditions,
                       uint8_t* alldata, detId_t* detId, stripId_t* stripId,
                       float *noise, float* gain, bool* bad)
{
  constexpr int nthreads = 128;
  const auto channels = chanlocs.size();
  const auto nblocks = (channels + nthreads - 1)/nthreads;
  
  unpackChannels<<<nblocks, nthreads>>>(chanlocs.onGPU(), conditions, alldata, detId, stripId, noise, gain, bad);
}

#endif
